#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <arpa/inet.h>
#include <netinet/in.h>
#include <sys/socket.h>
#include <unistd.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <nccl.h>
#include <mpi.h>

#define ITERS 100

#define SERVER_IP "10.10.2.1"
#define BASE_PORT 60071

static int g_mype = -1;

#define CUDA_CHECK_MSG(call, msg) \
    do { \
        hipError_t err__ = (call); \
        if (err__ != hipSuccess) { \
            fprintf(stderr, "[PE %d] CUDA error during %s at %s:%d on call %s: %s\n", \
                    g_mype, msg, __FILE__, __LINE__, #call, hipGetErrorString(err__)); \
            nvshmem_global_exit(1); \
        } \
    } while (0)

static inline void check_cuda_last_error(const char *api_name, const char *stage) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "[PE %d] CUDA kernel error for %s during %s: %s\n",
                g_mype, api_name ? api_name : "(unspecified)", stage, hipGetErrorString(err));
        nvshmem_global_exit(1);
    }
}

static inline void check_cuda_error(hipError_t err, const char *api_name, const char *action) {
    if (err != hipSuccess) {
        fprintf(stderr, "[PE %d] CUDA error for %s during %s: %s\n",
                g_mype, api_name ? api_name : "(unspecified)", action, hipGetErrorString(err));
        nvshmem_global_exit(1);
    }
}

// ================= KERNELS =================

__global__ void send_block(void *dest, void *src, int size, uint64_t *sig, int peer) {
    int num_blocks = gridDim.x;
    int size_per_block = size / num_blocks;
    int x = blockIdx.x;
    int mype = nvshmem_my_pe();
    nvshmem_signal_wait_until(sig, NVSHMEM_CMP_EQ, 0);
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        nvshmemx_signal_op(sig, 1, NVSHMEM_SIGNAL_SET, mype);
    }
    nvshmemx_putmem_signal_nbi_block(
        (char*)dest + x * size_per_block, (char*)src + x * size_per_block, size_per_block,
        &sig[mype], 1, NVSHMEM_SIGNAL_ADD, peer);
}

__global__ void wait_kernel(uint64_t *sig, int num_send_blocks) {
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    if (threadIdx.x > 0 && threadIdx.x < npes) {
        nvshmem_signal_wait_until(&sig[threadIdx.x], NVSHMEM_CMP_EQ, num_send_blocks);
        nvshmemx_signal_op(&sig[threadIdx.x], 0, NVSHMEM_SIGNAL_SET, mype);
        nvshmemx_signal_op(sig, 0, NVSHMEM_SIGNAL_SET, threadIdx.x);
    }
}

class Timer {
    private:
        hipEvent_t start_event, stop_event;
        hipStream_t stream;
        float ms;
    public:
        explicit Timer(hipStream_t s) : start_event(nullptr), stop_event(nullptr), stream(s), ms(0.0f) {
            CUDA_CHECK_MSG(hipEventCreate(&start_event), "hipEventCreate(start)");
            CUDA_CHECK_MSG(hipEventCreate(&stop_event), "hipEventCreate(stop)");
        }
        ~Timer() {
            hipEventDestroy(start_event);
            hipEventDestroy(stop_event);
        }
        void start() { CUDA_CHECK_MSG(hipEventRecord(start_event, stream), "hipEventRecord(start)"); }
        void stop() {
            CUDA_CHECK_MSG(hipEventRecord(stop_event, stream), "hipEventRecord(stop)");
            CUDA_CHECK_MSG(hipEventSynchronize(stop_event), "hipEventSynchronize(stop)");
            CUDA_CHECK_MSG(hipEventElapsedTime(&ms, start_event, stop_event), "hipEventElapsedTime");
        }
        float get_elapsed_time_ms() const { return ms; }
};

// ================= TCP helpers =================

static int tcp_listen_and_accept(int port) {
    int sockfd = ::socket(AF_INET, SOCK_STREAM, 0);
    if (sockfd < 0) { perror("socket"); return -1; }
    int yes = 1; setsockopt(sockfd, SOL_SOCKET, SO_REUSEADDR, &yes, sizeof(yes));
    sockaddr_in addr{}; addr.sin_family = AF_INET; addr.sin_port = htons(port);
    inet_pton(AF_INET, SERVER_IP, &addr.sin_addr);
    if (bind(sockfd, (sockaddr*)&addr, sizeof(addr)) != 0) { perror("bind"); close(sockfd); return -1; }
    if (listen(sockfd, 1) != 0) { perror("listen"); close(sockfd); return -1; }
    int conn = accept(sockfd, nullptr, nullptr);
    if (conn < 0) { perror("accept"); close(sockfd); return -1; }
    close(sockfd);
    return conn;
}

static int tcp_connect_retry(int port) {
    int sockfd = ::socket(AF_INET, SOCK_STREAM, 0);
    if (sockfd < 0) { perror("socket"); return -1; }
    sockaddr_in addr{}; addr.sin_family = AF_INET; addr.sin_port = htons(port);
    inet_pton(AF_INET, SERVER_IP, &addr.sin_addr);
    const int maxAttempts = 50;
    int attempt = 0;
    while (connect(sockfd, (sockaddr*)&addr, sizeof(addr)) != 0) {
        if (++attempt >= maxAttempts) { perror("connect"); close(sockfd); return -1; }
        usleep(100000);
    }
    return sockfd;
}

// ================= Bench: NVSHMEM (cross-node, MPI bootstrap) =================

static void bench_nvshmem(int msg_size, int num_blocks) {
    int mype = nvshmem_my_pe();
    g_mype = mype;
    int npes = nvshmem_n_pes();
    if (npes != 2) {
        if (mype == 0) fprintf(stderr, "NVSHMEM test requires exactly 2 PEs\n");
        nvshmem_barrier_all();
        return;
    }

    int buffer_size = msg_size * npes;
    int sig_bytes = npes * (int)sizeof(uint64_t);
    void *buf = nvshmem_malloc(sig_bytes + buffer_size);
    uint64_t *sig = (uint64_t*)buf;
    void *data = (char*)buf + sig_bytes;
    if (!buf) {
        fprintf(stderr, "[PE %d] nvshmem_malloc failed\n", mype);
        nvshmem_global_exit(1);
    }
    CUDA_CHECK_MSG(hipMemset((void*)sig, 0, sig_bytes), "hipMemset(sig)");

    void *d_src = nullptr;
    CUDA_CHECK_MSG(hipMalloc(&d_src, msg_size), "hipMalloc(d_src)");
    if (mype == 1) {
        int *h_src = (int*)malloc(msg_size);
        for (size_t i = 0; i < (size_t)msg_size / sizeof(int); i++) h_src[i] = 1;
        CUDA_CHECK_MSG(hipMemcpy(d_src, h_src, msg_size, hipMemcpyHostToDevice), "H2D d_src");
        free(h_src);
    }

    hipStream_t stream{}; CUDA_CHECK_MSG(hipStreamCreate(&stream), "hipStreamCreate");
    Timer timer(stream);

    if (mype == 1) {
        int num_threads = 256;
        void *dst_peer0 = (char*)data + mype * msg_size; // write to our slot on PE0's symmetric heap
        auto run_once = [&]() {
            send_block<<<num_blocks, num_threads, 0, stream>>>(dst_peer0, d_src, msg_size, sig, 0);
            check_cuda_last_error("send_block", "launch");
            check_cuda_error(hipStreamSynchronize(stream), "send_block", "sync");
        };
        for (int i = 0; i < 10; i++) run_once();
        nvshmem_barrier_all();
        timer.start();
        for (int i = 0; i < ITERS; i++) run_once();
        timer.stop();
        printf("[PE %d] NVSHMEM send avg %.3f us per iter\n", mype, (timer.get_elapsed_time_ms() * 1000.0f) / ITERS);
    } else {
        auto run_once = [&]() {
            wait_kernel<<<1, 32, 0, stream>>>(sig, num_blocks);
            check_cuda_last_error("wait_kernel", "launch");
            check_cuda_error(hipStreamSynchronize(stream), "wait_kernel", "sync");
        };
        for (int i = 0; i < 10; i++) run_once();
        nvshmem_barrier_all();
        timer.start();
        for (int i = 0; i < ITERS; i++) run_once();
        timer.stop();
        printf("[PE %d] NVSHMEM recv avg %.3f us per iter\n", mype, (timer.get_elapsed_time_ms() * 1000.0f) / ITERS);
    }

    hipStreamDestroy(stream);
    hipFree(d_src);
    nvshmem_free(buf);
}

// ================= Bench: NCCL (TCP rendezvous like latency-test) =================

static void bench_nccl(int msg_size, int mype) {
    int port = BASE_PORT + 1;
    int sockfd = -1;
    bool isServer = (mype == 0);
    if (isServer) sockfd = tcp_listen_and_accept(port);
    else sockfd = tcp_connect_retry(port);
    if (sockfd < 0) { fprintf(stderr, "[PE %d] TCP setup failed\n", mype); return; }

    setenv("NCCL_DEBUG", "WARN", 1);
    setenv("NCCL_SOCKET_IFNAME", "ibs3", 0);
    setenv("NCCL_IB_HCA", "mlx5", 0);
    setenv("NCCL_IB_DISABLE", "0", 0);

    int device = 0; CUDA_CHECK_MSG(hipSetDevice(device), "hipSetDevice(nccl)");

    size_t count_bytes = (size_t)msg_size;
    size_t numFloats = count_bytes / sizeof(float);
    if (numFloats == 0) numFloats = 1;
    float *d_send = nullptr, *d_recv = nullptr;
    CUDA_CHECK_MSG(hipMalloc(&d_send, numFloats * sizeof(float)), "hipMalloc d_send");
    CUDA_CHECK_MSG(hipMalloc(&d_recv, numFloats * sizeof(float)), "hipMalloc d_recv");
    CUDA_CHECK_MSG(hipMemset(d_recv, 0, numFloats * sizeof(float)), "memset d_recv");

    ncclUniqueId id{};
    if (isServer) {
        ncclGetUniqueId(&id);
        if (send(sockfd, &id, sizeof(id), 0) != (ssize_t)sizeof(id)) { perror("send id"); close(sockfd); return; }
    } else {
        if (recv(sockfd, &id, sizeof(id), MSG_WAITALL) != (ssize_t)sizeof(id)) { perror("recv id"); close(sockfd); return; }
    }

    ncclComm_t comm{};
    ncclCommInitRank(&comm, 2, id, mype);
    hipStream_t stream{}; CUDA_CHECK_MSG(hipStreamCreate(&stream), "hipStreamCreate");

    // Warmup
    ncclGroupStart();
    if (isServer) {
        ncclRecv(d_recv, numFloats, ncclFloat, 1, comm, stream);
        ncclSend(d_send, numFloats, ncclFloat, 1, comm, stream);
    } else {
        ncclSend(d_send, numFloats, ncclFloat, 0, comm, stream);
        ncclRecv(d_recv, numFloats, ncclFloat, 0, comm, stream);
    }
    ncclGroupEnd();
    CUDA_CHECK_MSG(hipStreamSynchronize(stream), "warmup sync");

    Timer timer(stream);
    timer.start();
    for (int i = 0; i < ITERS; i++) {
        ncclGroupStart();
        if (isServer) {
            ncclRecv(d_recv, numFloats, ncclFloat, 1, comm, stream);
        } else {
            ncclSend(d_send, numFloats, ncclFloat, 0, comm, stream);
        }
        ncclGroupEnd();
        CUDA_CHECK_MSG(hipStreamSynchronize(stream), "iter sync");
    }
    timer.stop();
    if (isServer) {
        printf("[PE %d] NCCL recv avg %.3f us per iter\n", mype, (timer.get_elapsed_time_ms() * 1000.0f) / ITERS);
    } else {
        printf("[PE %d] NCCL send avg %.3f us per iter\n", mype, (timer.get_elapsed_time_ms() * 1000.0f) / ITERS);
    }

    hipStreamDestroy(stream);
    ncclCommDestroy(comm);
    hipFree(d_send); hipFree(d_recv);
    close(sockfd);
}

int main(int argc, char **argv) {
    MPI_Init(&argc, &argv);

    int world_rank = 0, world_size = 1;
    MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);

    int local_rank = 0;
    {
        // Try to derive local rank from MPI, fallback to 0
        MPI_Comm shmcomm; MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, 0, MPI_INFO_NULL, &shmcomm);
        int shm_rank = 0; MPI_Comm_rank(shmcomm, &shm_rank);
        local_rank = shm_rank;
        MPI_Comm_free(&shmcomm);
    }

    int dev_count = 0; hipGetDeviceCount(&dev_count);
    if (dev_count == 0) { fprintf(stderr, "No CUDA devices\n"); MPI_Abort(MPI_COMM_WORLD, 1); }
    CUDA_CHECK_MSG(hipSetDevice(local_rank % dev_count), "hipSetDevice(local_rank)");

    nvshmemx_init_attr_t attr{};
    attr.mpi_comm = MPI_COMM_WORLD;
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);

    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    g_mype = mype;
    if (npes != 2) {
        if (mype == 0) fprintf(stderr, "Please run with exactly 2 MPI ranks across two hosts.\n");
        nvshmem_finalize();
        MPI_Finalize();
        return 1;
    }

    const int M = 1024 * 1024;
    const int sizes[] = { 1 * M, 2 * M, 4 * M, 8 * M };

    for (int i = 0; i < (int)(sizeof(sizes)/sizeof(sizes[0])); i++) {
        int sz = sizes[i];
        if (sz >= M) {
            printf("msg_size %d MB\n", sz / M);
        } else {
            printf("msg_size %d KB\n", sz / 1024);
        }
        // NVSHMEM benchmark (cross-node)
        bench_nvshmem(sz, /*num_blocks*/8);
        // NCCL benchmark (TCP rendezvous)
        bench_nccl(sz, mype);
        if (mype == 0) printf("--------------------------------\n");
        nvshmem_barrier_all();
    }

    nvshmem_finalize();
    MPI_Finalize();
    return 0;
}


